
#include <hip/hip_runtime.h>
///
/// vecAddKernel00.cu
/// For COMS E6998 Spring 2023
/// Instructor: Parajit Dube and Kaoutar El Maghraoui
/// Based on code from the CUDA Programming Guide
/// By David Newman
/// Created: 2011-02-16
/// Last Modified: 2011-02-16 DVN
///
/// This Kernel adds two Vectors A and B in C on GPU
/// without using coalesced memory access.
/// 

__global__ void AddVectorsModified(const float* A, const float* B, float* C, int N)
{
    int idx = (blockIdx.x * blockDim.x) + threadIdx.x; // Unique grid index of a thread
    if(idx < N) {
        C[idx] = A[idx] + B[idx];
    }
}

