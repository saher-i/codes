
#include <hip/hip_runtime.h>
///
/// vecAddKernel00.cu
/// For COMS E6998 Spring 2023
/// Instructor: Parajit Dube and Kaoutar El Maghraoui
/// Based on code from the CUDA Programming Guide
/// By David Newman
/// Created: 2011-02-16
/// Last Modified: 2011-02-16 DVN
///
/// This Kernel adds two Vectors A and B in C on GPU
/// without using coalesced memory access.
/// 

__global__ void AddVectors(const float* A, const float* B, float* C, int number_of_elements)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Unique grid index of a thread
    int stride = blockDim.x * gridDim.x; // Total number of threads in the grid

    for (int i = idx; i < number_of_elements; i += stride) {
        C[i] = A[i] + B[i];
    }
}

