
#include <hip/hip_runtime.h>
///
/// vecAddKernel00.cu
/// For COMS E6998 Spring 2023
/// Instructor: Parajit Dube and Kaoutar El Maghraoui
/// Based on code from the CUDA Programming Guide
/// By David Newman
/// Created: 2011-02-16
/// Last Modified: 2011-02-16 DVN
///
/// This Kernel adds two Vectors A and B in C on GPU
/// without using coalesced memory access.
///

__global__ void AddVectors(const float *A, const float *B, float *C, int N) {
  
    // Global thread index calculation
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    for(int i = index; i < blockDim.x*gridDim.x*N; i = i+N) {
        C[i] = A[i] + B[i];
    }
}
