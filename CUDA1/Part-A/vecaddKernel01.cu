
#include <hip/hip_runtime.h>
///
/// vecAddKernel00.cu
/// For COMS E6998 Spring 2023
/// Instructor: Parajit Dube and Kaoutar El Maghraoui
/// Based on code from the CUDA Programming Guide
/// By David Newman
/// Created: 2011-02-16
/// Last Modified: 2011-02-16 DVN
///
/// This Kernel adds two Vectors A and B in C on GPU
/// without using coalesced memory access.
/// 

__global__ void AddVectors(float* A, float* B, float* C, int N)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Unique grid index of a thread
    int stride = blockDim.x * gridDim.x; // Total number of threads in the grid

    for (int i = idx; i < N; i += stride) {
        B[i] = A[i] + B[i];
        C[i] = B[i];
    }
}

