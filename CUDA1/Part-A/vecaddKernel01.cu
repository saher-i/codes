#include "hip/hip_runtime.h"
///
/// vecAddKernel00.cu
/// For COMS E6998 Spring 2023
/// Instructor: Parajit Dube and Kaoutar El Maghraoui
/// Based on code from the CUDA Programming Guide
/// By David Newman
/// Created: 2011-02-16
/// Last Modified: 2011-02-16 DVN
///
/// This Kernel adds two Vectors A and B in C on GPU
/// without using coalesced memory access.
/// 

__global__ void AddVectors(const float* A, const float* B, float* C, int N)
{
    int startidx = ((blockIdx.x * blockDim.x) + threadIdx.x)*32; // Unique grid index of a thread
    if(startidx < N) {
        for (int j = 0; j < 32 && (startIdx + j) < N; ++j){
            int idx = startIdx + j;
            C[idx] = A[idx] + B[idx];
        }
    }
}

