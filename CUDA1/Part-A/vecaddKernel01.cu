#include "hip/hip_runtime.h"
///
/// vecAddKernel00.cu
/// For COMS E6998 Spring 2023
/// Instructor: Parajit Dube and Kaoutar El Maghraoui
/// Based on code from the CUDA Programming Guide
/// By David Newman
/// Created: 2011-02-16
/// Last Modified: 2011-02-16 DVN
///
/// This Kernel adds two Vectors A and B in C on GPU
/// without using coalesced memory access.
///

__global__ void AddVectors(const float *A, const float *B, float *C, int N) {
  
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    for(int i = 0; i < N; i+blockDim.x)) {
        C[index] = A[index] + B[index];
    }
/*
    for(int i = index; i < N; i = i+blockDim.x) {
        C[i] = A[i] + B[i];
    }

    for(int i = index; i < blockDim.x*gridDim.x*N; i = i+N) {
        C[i] = A[i] + B[i];
    }
*/
}
